#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "spatial.h"
#include "custom_knn.h"

std::tuple<torch::Tensor, torch::Tensor>
topKdistCUDA2(const torch::Tensor& points)
{
  const int P = points.size(0);

  auto float_opts = points.options().dtype(torch::kFloat32);
  auto int_opts = points.options().dtype(torch::kInt32);
  torch::Tensor topKDist = torch::full({P*8}, 0.0, float_opts);
  torch::Tensor topKIndices = torch::full({P*8}, 0.0, int_opts);
  
  CustomKNN::custom_knn(P, (float3*)points.contiguous().data<float>(), 
  topKDist.contiguous().data<float>(), topKIndices.contiguous().data<int>());

  return std::make_tuple(topKDist, topKIndices);
}